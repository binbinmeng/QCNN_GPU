#include "hip/hip_runtime.h"
﻿/*
#include <stdio.h>
#include <sstream>
#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>

#include <hipblas.h>
#include <hipDNN.h>

#include "yuv_data.h"

struct qvrcnn_data {
	char weight[5 * 5 * 64];
	int bias[64];
};//实验性，仅读取第一层

void HWCN2NHWC(char *HWCN, char *NHWC, int H, int W, int C, int N)
{
	int i, j, k, m;
	for (i = 0;i < H;i++)
		for (j = 0;j < W;j++)
			for (k = 0;k < C;k++)
				for (m = 0;m < N;m++)
					NHWC[m*H*W*C + i*W*C + j*C + k] = HWCN[i*W*C*N + j*C*N + k*N + m];
}

struct qvrcnn_data* read_qvrcnn(void)
{
	struct qvrcnn_data net_data_HWCN;
	struct qvrcnn_data* net_data_NHWC = new struct qvrcnn_data;
	FILE  *fp = NULL;
	if (fopen_s(&fp, "model\\qvrcnn_ppro_8bit_27.data", "rb"))
		printf("open file failed\n");
	fseek(fp, sizeof(int), SEEK_CUR);
	fread(net_data_HWCN.weight, sizeof(char), 5 * 5 * 64, fp);
	fread(net_data_NHWC->bias, sizeof(int), 64, fp);
	fclose(fp);
	HWCN2NHWC(net_data_HWCN.weight, net_data_NHWC->weight, 5, 5, 1, 64);//convert format
	return net_data_NHWC;
}

int main(int argc, char** argv)
{
	int num_gpus;
	hipblasHandle_t cublasHandle;
	hipdnnHandle_t cudnnHandle;
	hipdnnTensorDescriptor_t dataTensor, conv1Tensor, conv1BiasTensor;//数据和偏置描述符
	hipdnnFilterDescriptor_t conv1filterDesc;//权重描述符
	hipdnnConvolutionDescriptor_t conv1Desc;//卷积描述符
	hipdnnConvolutionFwdAlgoPerf_t perfResults[8];
	size_t sizeInBytes;

	YChannel *ydata;
	Res *ydata_reg;
	qvrcnn_data* net_data;
	int batch = 1, channel = 0, height = 240, width = 416, return_value, return_value1;
	std::stringstream filename;

	hipGetDeviceCount(&num_gpus);
	hipSetDevice(0);
	hipblasCreate(&cublasHandle);
	hipdnnCreate(&cudnnHandle);
	hipdnnCreateTensorDescriptor(&dataTensor);//初始化张量描述符
	hipdnnCreateTensorDescriptor(&conv1Tensor);
	hipdnnCreateTensorDescriptor(&conv1BiasTensor);

	hipdnnCreateFilterDescriptor(&conv1filterDesc);//初始化权重描述符

	hipdnnCreateConvolutionDescriptor(&conv1Desc);//初始化卷积描述符

												 //设置卷积描述符
	return_value = hipdnnSetTensor4dDescriptor(conv1BiasTensor,
		HIPDNN_TENSOR_NHWC,
		HIPDNN_DATA_INT32,
		1, 64, 1, 1);
	
	return_value = hipdnnSetTensor4dDescriptor(dataTensor,
		HIPDNN_TENSOR_NHWC,
		HIPDNN_DATA_INT8,
		1, 1, height, width);
	return_value = hipdnnSetFilter4dDescriptor(conv1filterDesc,
		HIPDNN_DATA_INT8,
		HIPDNN_TENSOR_NHWC,
		64, 1, 5, 5);
	return_value = hipdnnSetConvolution2dDescriptor(conv1Desc,
		2, 2,
		1, 1,
		1, 1,
		HIPDNN_CROSS_CORRELATION,
		HIPDNN_DATA_INT32);
	return_value = hipdnnGetConvolution2dForwardOutputDim(conv1Desc,
		dataTensor,
		conv1filterDesc,
		&batch, &channel, &height, &width);
	return_value = hipdnnSetTensor4dDescriptor(conv1Tensor,
		HIPDNN_TENSOR_NHWC,
		HIPDNN_DATA_FLOAT,
		1, 64, height, width);
	cudnnGetConvolutionForwardAlgorithmMaxCount(cudnnHandle, &return_value);
	return_value = cudnnGetConvolutionForwardAlgorithm_v7(cudnnHandle,
		dataTensor,
		conv1filterDesc,
		conv1Desc,
		conv1Tensor,
		8,
		&return_value1,
		perfResults);
	return_value = hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
		dataTensor,
		conv1filterDesc,
		conv1Desc,
		conv1Tensor,
		perfResults[0].algo,
		&sizeInBytes);

	char *x;
	float *conv,*conv_h;//前向传播数据
	char *w;
	int *b;//网络参数
	void *d_cudnn_workspace = nullptr;//缓存和工作空间
	float alpha = 1.0f, beta = 0.0f;
	clock_t start_t, end_t;
	double total_t;
	int i;
	//读取网络和数据
	net_data = read_qvrcnn();
	filename << "data\\BlowingBubbles_intra_main_HM16.7_anchor_416x240_10_Q27.yuv";
	ydata = get_Y(filename.str().c_str(), batch, height, width);
	ydata_reg = regularize(ydata);

	return_value = hipMalloc(&x, sizeof(char) * 416 * 240);//在GPU中分配空间
	return_value = hipMalloc(&w, sizeof(char) * 5 * 5 * 64);
	return_value = hipMalloc(&b, sizeof(int) * 64);
	return_value = hipMalloc(&conv, sizeof(float) * 64 * 416 * 240);
	conv_h = (float*)malloc(sizeof(float) * 64 * 416 * 240);
	if (sizeInBytes > 0)
		return_value = hipMalloc(&d_cudnn_workspace, sizeInBytes);//分配工作空间

	return_value = hipMemcpyAsync(w, net_data->weight, sizeof(char) * 5 * 5 * 64, hipMemcpyHostToDevice);//拷贝网络到GPU
	return_value = hipMemcpyAsync(x, ydata_reg->data,sizeof(char) * ydata->frames*ydata->h*ydata->w, hipMemcpyHostToDevice);//拷贝数据到GPU

	start_t = clock();
	for (i = 0;i < 10000;i++)
	{
		return_value = hipdnnConvolutionForward(cudnnHandle, &alpha, dataTensor,
			x, conv1filterDesc, w, conv1Desc,
			perfResults[0].algo, d_cudnn_workspace, sizeInBytes, &beta,
			conv1Tensor, conv);//进行一次卷积运算
		return_value = hipDeviceSynchronize();//同步GPU
	}
	end_t = clock();
	total_t = (double)(end_t - start_t) / CLOCKS_PER_SEC;
	printf("%f", total_t);
	return_value = hipMemcpy(conv_h, conv, sizeof(float) * 64 * 416 * 240, hipMemcpyDeviceToHost);
	//到此步即可完成debug						
	//hipMemcpy(&conv1.pconv[0], d_pconv1, sizeof(float) * conv1.pconv.size(), hipMemcpyDeviceToHost);//从GPU中拷贝出数据
	return_value = hipFree(x);//释放内存
	return_value = hipFree(w);
	return_value = hipFree(b);
	return_value = hipFree(conv);
	return_value = hipFree(d_cudnn_workspace);
	system("pause");
	return 0;
}
*/
#include <stdio.h>
#include <sstream>
#include <iostream>
#include <time.h>
#include <hip/hip_runtime.h>
#include <>

#include <hipblas.h>
#include <hipDNN.h>

#include "yuv_data.h"

struct vrcnn_data {
	float weight[5 * 5 * 64];
	float bias[64];
};//实验性，仅读取第一层

void HWCN2NHWC(float *HWCN, float *NHWC, int H, int W, int C, int N)
{
	int i, j, k, m;
	for (i = 0;i < H;i++)
		for (j = 0;j < W;j++)
			for (k = 0;k < C;k++)
				for (m = 0;m < N;m++)
					NHWC[m*H*W*C + i*W*C + j*C + k] = HWCN[i*W*C*N + j*C*N + k*N + m];
}

struct vrcnn_data* read_vrcnn(void)
{
	struct vrcnn_data net_data_HWCN;
	struct vrcnn_data* net_data_NHWC = new struct vrcnn_data;
	FILE  *fp = NULL;
	if (fopen_s(&fp, "model\\vrcnn_ppro_27.data", "rb"))
		printf("open file failed\n");
	fread(net_data_HWCN.weight, sizeof(float), 5 * 5 * 64, fp);
	fread(net_data_NHWC->bias, sizeof(float), 64, fp);
	fclose(fp);
	HWCN2NHWC(net_data_HWCN.weight, net_data_NHWC->weight, 5, 5, 1, 64);//convert format
	return net_data_NHWC;
}
float *regularizef(YChannel *ydata)
{
	int i;
	float *reg = (float*)malloc(sizeof(float)*ydata->frames*ydata->h*ydata->w);
	for (i = 0; i < ydata->frames*ydata->h*ydata->w; i++)
		reg[i] = (int)ydata->ImgData[i] - 128;
	return reg;

}
int main(int argc, char** argv)
{
	int num_gpus;
	hipblasHandle_t cublasHandle;
	hipdnnHandle_t cudnnHandle;
	hipdnnTensorDescriptor_t dataTensor, conv1Tensor, conv1BiasTensor;//数据和偏置描述符
	hipdnnFilterDescriptor_t conv1filterDesc;//权重描述符
	hipdnnConvolutionDescriptor_t conv1Desc;//卷积描述符
	hipdnnConvolutionFwdAlgoPerf_t perfResults[8];
	size_t sizeInBytes;

	YChannel *ydata;
	float *ydata_reg;
	vrcnn_data* net_data;
	int batch = 1, channel = 0, height = 240, width = 416, return_value, return_value1;
	std::stringstream filename;

	hipGetDeviceCount(&num_gpus);
	hipSetDevice(0);
	hipblasCreate(&cublasHandle);
	hipdnnCreate(&cudnnHandle);
	hipdnnCreateTensorDescriptor(&dataTensor);//初始化张量描述符
	hipdnnCreateTensorDescriptor(&conv1Tensor);
	hipdnnCreateTensorDescriptor(&conv1BiasTensor);

	hipdnnCreateFilterDescriptor(&conv1filterDesc);//初始化权重描述符

	hipdnnCreateConvolutionDescriptor(&conv1Desc);//初始化卷积描述符

												 //设置卷积描述符
	return_value = hipdnnSetTensor4dDescriptor(conv1BiasTensor,
		HIPDNN_TENSOR_NHWC,
		HIPDNN_DATA_FLOAT,
		1, 64, 1, 1);

	return_value = hipdnnSetTensor4dDescriptor(dataTensor,
		HIPDNN_TENSOR_NHWC,
		HIPDNN_DATA_FLOAT,
		1, 1, height, width);
	return_value = hipdnnSetFilter4dDescriptor(conv1filterDesc,
		HIPDNN_DATA_FLOAT,
		HIPDNN_TENSOR_NHWC,
		64, 1, 5, 5);
	return_value = hipdnnSetConvolution2dDescriptor(conv1Desc,
		2, 2,
		1, 1,
		1, 1,
		HIPDNN_CROSS_CORRELATION,
		HIPDNN_DATA_FLOAT);
	return_value = hipdnnGetConvolution2dForwardOutputDim(conv1Desc,
		dataTensor,
		conv1filterDesc,
		&batch, &channel, &height, &width);
	return_value = hipdnnSetTensor4dDescriptor(conv1Tensor,
		HIPDNN_TENSOR_NHWC,
		HIPDNN_DATA_FLOAT,
		1, 64, height, width);
	cudnnGetConvolutionForwardAlgorithmMaxCount(cudnnHandle, &return_value);
	return_value = cudnnGetConvolutionForwardAlgorithm_v7(cudnnHandle,
		dataTensor,
		conv1filterDesc,
		conv1Desc,
		conv1Tensor,
		8,
		&return_value1,
		perfResults);
	return_value = hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
		dataTensor,
		conv1filterDesc,
		conv1Desc,
		conv1Tensor,
		perfResults[0].algo,
		&sizeInBytes);

	char *x;
	float *conv, *conv_h;//前向传播数据
	float *w;
	int *b;//网络参数
	void *d_cudnn_workspace = nullptr;//缓存和工作空间
	float alpha = 1.0f, beta = 0.0f;
	clock_t start_t, end_t;
	double total_t;
	int i;
	//读取网络和数据
	net_data = read_vrcnn();
	filename << "data\\BlowingBubbles_intra_main_HM16.7_anchor_416x240_10_Q27.yuv";
	ydata = get_Y(filename.str().c_str(), batch, height, width);
	ydata_reg = regularizef(ydata);

	return_value = hipMalloc(&x, sizeof(float) * 416 * 240);//在GPU中分配空间
	return_value = hipMalloc(&w, sizeof(float) * 5 * 5 * 64);
	return_value = hipMalloc(&b, sizeof(float) * 64);
	return_value = hipMalloc(&conv, sizeof(float) * 64 * 416 * 240);
	conv_h = (float*)malloc(sizeof(float) * 64 * 416 * 240);
	if (sizeInBytes > 0)
		return_value = hipMalloc(&d_cudnn_workspace, sizeInBytes);//分配工作空间

	return_value = hipMemcpyAsync(w, net_data->weight, sizeof(float) * 5 * 5 * 64, hipMemcpyHostToDevice);//拷贝网络到GPU
	return_value = hipMemcpyAsync(x, ydata_reg, sizeof(float) * ydata->frames*ydata->h*ydata->w, hipMemcpyHostToDevice);//拷贝数据到GPU

	start_t = clock();
	for (i = 0;i < 10000;i++)
	{
		return_value = hipdnnConvolutionForward(cudnnHandle, &alpha, dataTensor,
			x, conv1filterDesc, w, conv1Desc,
			perfResults[0].algo, d_cudnn_workspace, sizeInBytes, &beta,
			conv1Tensor, conv);//进行一次卷积运算
		return_value = hipDeviceSynchronize();//同步GPU
	}
	end_t = clock();
	total_t = (double)(end_t - start_t) / CLOCKS_PER_SEC;
	printf("%f", total_t);
	return_value = hipMemcpy(conv_h, conv, sizeof(float) * 64 * 416 * 240, hipMemcpyDeviceToHost);
	//到此步即可完成debug						
	//hipMemcpy(&conv1.pconv[0], d_pconv1, sizeof(float) * conv1.pconv.size(), hipMemcpyDeviceToHost);//从GPU中拷贝出数据
	return_value = hipFree(x);//释放内存
	return_value = hipFree(w);
	return_value = hipFree(b);
	return_value = hipFree(conv);
	return_value = hipFree(d_cudnn_workspace);
	system("pause");
	return 0;
}