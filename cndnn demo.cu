#include <stdio.h>
#include <sstream>
#include <iostream>

#include <hip/hip_runtime.h>
#include <>

#include <hipblas.h>
#include <hipDNN.h>

#include "yuv_data.h"

struct qvrcnn_data {
	char weight[5*5*64];
	char bias[64];
};//实验性，仅读取第一层

struct qvrcnn_data* read_qvrcnn(void)
{
	struct qvrcnn_data* net_data = new struct qvrcnn_data;
	FILE  *fp = NULL;
	if (fopen_s(&fp, "model\\qvrcnn_8bit_22.data", "rb"))
		printf("open file failed\n");
		fread(net_data->weight, sizeof(char), 5*5*64, fp);
	fclose(fp);
	return net_data;
}

int main(int argc, char** argv)
{
	hipblasHandle_t cublasHandle;
	hipdnnHandle_t cudnnHandle;
	hipdnnTensorDescriptor_t dataTensor, conv1Tensor, conv1BiasTensor;//数据和偏置描述符
	hipdnnFilterDescriptor_t conv1filterDesc;//权重描述符
	hipdnnConvolutionDescriptor_t conv1Desc;//卷积描述符
	//hipdnnConvolutionFwdAlgo_t conv1algo;//卷积算法描述符
	hipdnnConvolutionFwdAlgoPerf_t perfResults[8];
	size_t sizeInBytes;

	YChannel *ydata, *conv1_out;
	qvrcnn_data* net_data;
	int batch = 1, channel = 0, height = 240, width = 416, return_value, return_value1;
	std::stringstream filename;
	filename << "data\\BlowingBubbles_intra_main_HM16.7_anchor_416x240_10_Q22.yuv";
	ydata = get_Y(filename.str().c_str(), batch, height, width);
	conv1_out = get_Y(NULL, 64, height, width);

	int num_gpus;
	hipGetDeviceCount(&num_gpus);
	hipSetDevice(0);
	hipblasCreate(&cublasHandle);
	hipdnnCreate(&cudnnHandle);
	hipdnnCreateTensorDescriptor(&dataTensor);//初始化张量描述符
	hipdnnCreateTensorDescriptor(&conv1Tensor);
	hipdnnCreateTensorDescriptor(&conv1BiasTensor);

	hipdnnCreateFilterDescriptor(&conv1filterDesc);//初始化权重描述符

	hipdnnCreateConvolutionDescriptor(&conv1Desc);//初始化卷积描述符
	
	//设置卷积描述符
	return_value = hipdnnSetTensor4dDescriptor(conv1BiasTensor,
		HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_INT32,
		batch, 64, 1, 1);

	return_value = hipdnnSetTensor4dDescriptor(dataTensor,
		HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_INT32,
		1, 1, height, width);
	return_value = hipdnnSetFilter4dDescriptor(conv1filterDesc,
		HIPDNN_DATA_INT32,
		HIPDNN_TENSOR_NCHW,
		64, 1, 5, 5);
	return_value = hipdnnSetConvolution2dDescriptor(conv1Desc,
		2, 2,
		1, 1,
		1, 1,
		HIPDNN_CROSS_CORRELATION,
		HIPDNN_DATA_INT32);
	return_value = hipdnnGetConvolution2dForwardOutputDim(conv1Desc,
		dataTensor,
		conv1filterDesc,
		&batch, &channel, &height, &width);
	return_value = hipdnnSetTensor4dDescriptor(conv1Tensor,
		HIPDNN_TENSOR_NCHW,
		HIPDNN_DATA_INT32,
		1, 64, height, width);
	cudnnGetConvolutionForwardAlgorithmMaxCount(cudnnHandle, &return_value);
	return_value = cudnnGetConvolutionForwardAlgorithm_v7(cudnnHandle,
		dataTensor,
		conv1filterDesc,
		conv1Desc,
		conv1Tensor,
		8,
		&return_value1,
		perfResults);
	return_value = hipdnnGetConvolutionForwardWorkspaceSize(cudnnHandle,
		dataTensor,
		conv1filterDesc,
		conv1Desc,
		conv1Tensor,
		perfResults[0].algo,
		&sizeInBytes);

	//读取或初始化网络权重
	net_data = read_qvrcnn();

	char *d_data;
	int *d_conv1;//前向传播数据
	float alpha = 1.0f, beta = 0.0f;
	return_value = hipMalloc(&d_data, sizeof(char) *1*416*240);//在GPU中分配空间
	return_value = hipMalloc(&d_conv1, sizeof(char) *64*416*240);

	char *d_pconv1, *d_pconv1bias;//网络参数
	return_value = hipMalloc(&d_pconv1, sizeof(char) * 5*5*64);
	return_value = hipMalloc(&d_pconv1bias, sizeof(char) * 64);
	
	void *d_cudnn_workspace = nullptr;//缓存和工作空间
	if (sizeInBytes > 0)
		return_value = hipMalloc(&d_cudnn_workspace, sizeInBytes);//分配工作空间

	return_value = hipMemcpyAsync(d_pconv1, net_data->weight, sizeof(char) * 5*5*64, hipMemcpyHostToDevice);//拷贝网络到GPU
	return_value = hipDeviceSynchronize();//同步GPU
	return_value = hipMemcpyAsync(d_data, ydata->ImgData,
		sizeof(char) * ydata->frames*ydata->h*ydata->w, hipMemcpyHostToDevice);//拷贝数据到GPU

	return_value = hipdnnConvolutionForward(cudnnHandle, &alpha, dataTensor,
		d_data, conv1filterDesc, d_pconv1, conv1Desc,
		perfResults[0].algo, d_cudnn_workspace, sizeInBytes, &beta,
		conv1Tensor, d_conv1);//进行一次卷积运算
	return_value = hipDeviceSynchronize();//同步GPU

	return_value = hipMemcpy(conv1_out->ImgData, d_conv1, sizeof(char) * 64 * 416 * 240, hipMemcpyDeviceToHost);
	//到此步即可完成debug						
	//hipMemcpy(&conv1.pconv[0], d_pconv1, sizeof(float) * conv1.pconv.size(), hipMemcpyDeviceToHost);//从GPU中拷贝出数据
	return_value = hipFree(d_data);//释放内存
	return_value = hipFree(d_conv1);
	return_value = hipFree(d_pconv1);
	return_value = hipFree(d_pconv1bias);
	return_value = hipFree(d_cudnn_workspace);

	return 0;
}